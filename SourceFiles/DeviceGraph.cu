#include "hip/hip_runtime.h"
#include "../Headers/DeviceGraph.cuh"
#include "hip/hip_runtime.h"
#include "../Headers/Common.cuh"

__host__ void indirectMalloc(void **ptr, int size) {
	void *p;
	hipMalloc(&p, size);
	hipMemcpy(ptr, &p, sizeof(void*), hipMemcpyHostToDevice);
}

__host__ DeviceGraph* loadGraphToDevice(const Graph *g) {
	DeviceGraph *res;
	hipMalloc((void**)&res, sizeof(DeviceGraph));
	indirectMalloc((void**)&res->vertices, g->vertexAmount*sizeof(DeviceGraphVertex));
	for (int i = 0; i < g->vertexAmount; i++) {
		DeviceGraphVertex v;
		v.neighbors = new int[g->vertices[i].neighbourhood.size()];
		int *neighbors;
		hipMalloc((void**)&neighbors, g->vertices[i].neighbourhood.size()*sizeof(int));
		for (unsigned int j = 0; j < g->vertices[i].neighbourhood.size(); j++) {
			v.neighbors[j] = g->vertices[i].neighbourhood[j];
		}
		v.degree = g->vertices[i].neighbourhood.size();
		hipMemcpy(neighbors, v.neighbors, g->vertices[i].neighbourhood.size()*sizeof(int), hipMemcpyHostToDevice);
		delete[] v.neighbors;
		v.neighbors = neighbors;
		hipMemcpy((&res->vertices)+i*sizeof(DeviceGraphVertex), &v, sizeof(DeviceGraphVertex), hipMemcpyHostToDevice);
	}
	hipMemcpy(&res->n, &g->vertexAmount, sizeof(int), hipMemcpyHostToDevice);
	return res;
}

__device__ bool DeviceBitset::operator[](int n) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = 0x1 << rem;
	return (contents[idx] & mask) ? true : false;
}

__device__ void DeviceBitset::set(int n, bool v) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = (v ? 1 : 0) << rem;
	contents[idx] = contents[idx] | mask;
}

__host__ DeviceBitset* createDeviceBitset(int n) {
	DeviceBitset *res;
	hipMalloc((void**)&res, sizeof(DeviceBitset));
	int idx = n / 8;
	indirectMalloc((void**)&res->contents, sizeof(char)*idx);
	for (int i = 0; i < idx; i++) res->contents[i] = (signed char)0xFF;
	hipMemcpy(&res->n, &n, sizeof(int), hipMemcpyHostToDevice);
	return res;
}

__host__ DeviceBitset** createBitsetArray(int n) {
	DeviceBitset **res;
	hipMalloc((void**)&res, n*sizeof(DeviceBitset*));
	for (int i = 0; i < n; i++) {
		DeviceBitset *temp = createDeviceBitset(n);
		hipMemcpy((&res) + i*sizeof(DeviceBitset*), &temp, sizeof(DeviceBitset*), hipMemcpyHostToDevice);
	}
	return res;
}

__host__ void unloadDeviceGraph(DeviceGraph *g) {
	for (int i = 0; i < g->n; i++)
		hipFree(g->vertices[i].neighbors);
	hipFree(g);
}
__host__ void freeDeviceBitset(DeviceBitset *b) {
	hipFree(b->contents);
	hipFree(b);
}

__host__ void freeBitsetArray(DeviceBitset **arr, int n) {
	for (int i = 0; i < n; i++) hipFree(arr+i*sizeof(DeviceBitset*));
	hipFree(arr);
}

__global__ void getWorthCudaKernel(DeviceBKInput **roadmap) {
	int myId = threadIdx.x;
	DeviceBKInput *myInput = roadmap[myId];
	getWorthDev(myInput);
}

__host__ void getWorthWithCuda(std::vector<Organism> &pop, DeviceGraph *g) {
	DeviceBKInput **map, **hostCopy = new DeviceBKInput*[pop.size()];
	std::vector<void*> general, bitsets, results;
	hipMalloc((void**)&map, sizeof(DeviceBKInput*)*pop.size());
	general.push_back(map);
	for (unsigned int i = 0; i < pop.size(); i++) {
		DeviceBKInput *in;
		hipMalloc((void**)&in, sizeof(DeviceBKInput));
		general.push_back(in);
		int *vertexMap;
		hipMalloc((void**)&vertexMap, pop[i].vertices.size()*sizeof(int));
		general.push_back(vertexMap);
		int *tempArray = new int[pop[i].vertices.size()], j=0;
		for (auto &t : pop[i].vertices) tempArray[j++] = t;
		hipMemcpy(vertexMap, tempArray, sizeof(int)*pop[i].vertices.size(), hipMemcpyHostToDevice);
		delete[] tempArray;
		hipMemcpy(&in->map, &vertexMap, sizeof(vertexMap), hipMemcpyHostToDevice);
		DeviceBitset** t = createBitsetArray(pop[i].vertices.size());
		bitsets.push_back(t);
		results.push_back(&in->result);
		hipMemcpy(&in->set, &t, sizeof(t), hipMemcpyHostToDevice);
		hipMemcpy(&in->g, &g, sizeof(g), hipMemcpyHostToDevice);
		hipMemcpy(&map[i], &in, sizeof(in), hipMemcpyHostToDevice);
	}
	//input array ready
	//time to launch CUDA kernel
	int n = pop.size();
	getWorthCudaKernel<<<1, n>>>(map);
	hipDeviceSynchronize(); //must remember this, or bad things will happen. baad things
	//cuda kernel launched and finished
	for (int i = 0; i < pop.size(); i++) {
		int temp;
		hipMemcpy(&temp, results[i], sizeof(int), hipMemcpyDeviceToHost);
		pop[i].worth = temp;
		for (auto &t : bitsets) freeBitsetArray((DeviceBitset**)t, pop[i].vertices.size());
	}
	for (auto &t : general) hipFree(t);
}