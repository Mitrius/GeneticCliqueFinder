#include "hip/hip_runtime.h"
#include "../Headers/DeviceGraph.cuh"
#include "hip/hip_runtime.h"

__host__ __device__ int RyBKA(DeviceBitset *stack, int *map, int *rsstack, int N, const DeviceGraph *graph) {
	int stackIdx = 1, cmax = -1;
	while (stackIdx > 0) {//while stack not empty
		stackIdx--; //stack pop
		if (stack[stackIdx].n == 0) {//if the P set is empty
			if (cmax < rsstack[stackIdx]) cmax = rsstack[stackIdx]; //check if found clique was greater than the previous one, if so, set
		}
		else {//if P is not empty
			int i = 0;//pick a vertex v
			while (!stack[stackIdx][i]) i++;//that exists in P
			//now to copy to unofficial N-th bitset
			for (int j = 0; j < (N / 8) + 1; j++) stack[N].contents[j] = stack[stackIdx].contents[j];
			rsstack[N] = rsstack[stackIdx];
			stack[N].n = stack[stackIdx].n;
			for (int j = 0; j < N; j++) {//push (P \ v)
				if (stack[N][j] && j != i) stack[stackIdx].set(i, 1);
				else stack[stackIdx].set(i, 0);
			}
			rsstack[stackIdx] = rsstack[N];
			stack[stackIdx].n = stack[N].n - 1;
			stackIdx++;
			int m = 0;
			for (int j = 0; j < N; j++) {//for every other vertex
				if (stack[N][j]) {//that exists, do
					if (graph->isEdge(map[i], map[j])) {//check for edge by the way of map
						stack[stackIdx].set(j, 1); //if connected, add to next iteration
						m++; //also, count
					}
					else stack[stackIdx].set(j, 0); //if not, make sure it won't be there.
				}

			}
			stack[stackIdx].n = m;
			rsstack[stackIdx] = rsstack[N] + 1;
			stackIdx++;
		}
	}
	return cmax;
}

__host__ DeviceGraph* loadGraphToDevice(const Graph *g) {
	DeviceGraph *res;
	hipMallocManaged(&res, sizeof(DeviceGraph));
	res->n = g->vertexAmount;
	hipMallocManaged(&res->vertices, g->vertexAmount*sizeof(DeviceGraphVertex));
	for (int i = 0; i < g->vertexAmount; i++) {
		int m = g->vertices[i].neighbourhood.size();
		res->vertices[i].degree = m;
		hipMallocManaged(&res->vertices[i].neighbors, m*sizeof(int));
		for (unsigned int j = 0; j < m; j++) {
			res->vertices[i].neighbors[j] = g->vertices[i].neighbourhood[j];
		}
	}
	return res;
}

__host__ __device__ bool DeviceBitset::operator[](int n) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = 0x1 << rem;
	return (contents[idx] & mask) ? true : false;
}

__host__ __device__ void DeviceBitset::set(int n, bool v) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = 1 << rem;
	if (v) {
		if (contents[idx] != contents[idx] | mask) n++;
		contents[idx] = contents[idx] | mask;
	}
	else {
		if (contents[idx] != contents[idx] & ~mask) n--;
		contents[idx] = contents[idx] & ~mask;
	}
}

__host__ void unloadDeviceGraph(DeviceGraph *g) {
	for (int i = 0; i < g->n; i++) hipFree(g->vertices[i].neighbors);
	hipFree(g->vertices);
	hipFree(g);
}

__host__ DeviceBitset* createBitsetArray(int n) {
	DeviceBitset *res;
	hipMallocManaged(&res, (n+1)*sizeof(DeviceBitset));
	char *c;
	for (int i = 0; i < n+1; i++) {
		hipMallocManaged(&c, (n / 8) + 1);
		for (int j = 0; j < (n / 8) + 1; j++) c[j] = 0xFF;
		res[i].contents = c;
		res[i].n = n;
	}
	return res;
}

__global__ void getWorthCudaKernel(DeviceBKInput **roadmap) {
	int myId = threadIdx.x;
	DeviceBKInput* in = roadmap[myId];
	in->result = RyBKA(in->set, in->map, in->rsstack, in->set[0].n, in->g);
}

__host__ void getWorthWithCuda(std::vector<Organism> &pop, DeviceGraph *g) {
	int N = pop.size();
	DeviceBKInput **roadmap;
	hipMallocManaged(&roadmap, N*sizeof(DeviceBKInput*));
	for (int i = 0; i < N; i++) {
		int M = pop[i].vertices.size();
		DeviceBKInput *current;
		hipMallocManaged(&current, sizeof(DeviceBKInput));
		current->result = -1;
		current->g = g;
		hipMallocManaged(&current->map, (M+1)*sizeof(int));
		int j = 0;
		for (auto &t : pop[i].vertices) current->map[j++] = t;
		hipMallocManaged(&current->rsstack, M*sizeof(int));
		for (int k = 0; k < M; k++) current->rsstack[k] = 0;
		current->set = createBitsetArray(M);
		roadmap[i] = current;
	}
	for (int i = 0; i < N; i++){
		DeviceBKInput* in = roadmap[i];	
		in->result = RyBKA(in->set, in->map, in->rsstack, in->set[0].n, in->g);
	}
	for (int i = 0; i < N; i++) {
		pop[i].worth = roadmap[i]->result;
		hipFree(roadmap[i]->rsstack);
		hipFree(roadmap[i]->map);
		for (int j = 0; j < roadmap[i]->set->n; j++) hipFree(roadmap[i]->set[j].contents);
		hipFree(roadmap[i]->set);
		hipFree(roadmap[i]);
	}
	hipFree(roadmap);
}