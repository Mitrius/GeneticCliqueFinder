#include "hip/hip_runtime.h"
#include "../Headers/DeviceGraph.cuh"
#include "hip/hip_runtime.h"
#include "../Headers/Common.cuh"

__host__ DeviceGraph* loadGraphToDevice(const Graph *g) {
	DeviceGraph *res;
	hipMalloc((void**)&res, sizeof(DeviceGraph));
	hipMalloc((void**)&res->vertices, g->vertexAmount*sizeof(DeviceGraphVertex));
	for (int i = 0; i < g->vertexAmount; i++) {
		hipMalloc((void**)&res->vertices[i].neighbors, g->vertices[i].neighbourhood.size()*sizeof(int));
		for (unsigned int j = 0; j < g->vertices[i].neighbourhood.size(); j++) {
			res->vertices[i].neighbors[j] = g->vertices[i].neighbourhood[j];
		}
		res->vertices[i].degree = g->vertices[i].neighbourhood.size();
	}
	res->n = g->vertexAmount;
	return res;
}

__device__ bool DeviceBitset::operator[](int n) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = 0x1 << rem;
	return (contents[idx] & mask) ? true : false;
}

__device__ void DeviceBitset::set(int n, bool v) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = (v ? 1 : 0) << rem;
	contents[idx] = contents[idx] | mask;
}

__host__ DeviceBitset* createDeviceBitset(int n) {
	DeviceBitset *res;
	hipMalloc((void**)&res, sizeof(DeviceBitset));
	int idx = n / 8;
	hipMalloc((void**)&res->contents, sizeof(char)*idx);
	for (int i = 0; i < idx; i++) res->contents[i] = (signed char)0xFF;
	res->n = n;
	return res;
}

__host__ DeviceBitset** createBitsetArray(int n) {
	DeviceBitset **res;
	hipMalloc((void**)&res, n*sizeof(DeviceBitset*));
	for (int i = 0; i < n; i++) res[i] = createDeviceBitset(n);
	return res;
}

__host__ void unloadDeviceGraph(DeviceGraph *g) {
	for (int i = 0; i < g->n; i++)
		hipFree(g->vertices[i].neighbors);
	hipFree(g);
}
__host__ void freeDeviceBitset(DeviceBitset *b) {
	hipFree(b->contents);
	hipFree(b);
}

__host__ void freeBitsetArray(DeviceBitset **arr, int n) {
	for (int i = 0; i < n; i++) hipFree(arr[i]);
	hipFree(arr);
}

__global__ void getWorthCudaKernel(DeviceBKInput **roadmap) {
	int myId = threadIdx.x;
	DeviceBKInput *myInput = roadmap[myId];
	getWorthDev(myInput);
}

__host__ void getWorthWithCuda(std::vector<Organism> &pop, DeviceGraph *g) {
	DeviceBKInput **map, **hostCopy = new DeviceBKInput*[pop.size()];
	std::vector<void*> general, bitsets;
	hipMalloc((void**)&map, sizeof(DeviceBKInput*)*pop.size());
	general.push_back(map);
	for (unsigned int i = 0; i < pop.size(); i++) {
		DeviceBKInput *in;
		hipMalloc((void**)&in, sizeof(DeviceBKInput));
		general.push_back(in);
		int *vertexMap;
		hipMalloc((void**)&vertexMap, pop[i].vertices.size()*sizeof(int));
		general.push_back(vertexMap);
		int *tempArray = new int[pop[i].vertices.size()], j=0;
		for (auto &t : pop[i].vertices) tempArray[j++] = t;
		hipMemcpy(vertexMap, tempArray, sizeof(int)*pop[i].vertices.size(), hipMemcpyHostToDevice);
		delete[] tempArray;
		hipMemcpy(&in->map, &vertexMap, sizeof(vertexMap), hipMemcpyHostToDevice);
		DeviceBitset** t = createBitsetArray(pop[i].vertices.size());
		bitsets.push_back(t);
		hipMemcpy(&in->set, &t, sizeof(t), hipMemcpyHostToDevice);
		hipMemcpy(&in->g, &g, sizeof(g), hipMemcpyHostToDevice);
		hipMemcpy(&map[i], &in, sizeof(in), hipMemcpyHostToDevice);
	}
	//input array ready
	//time to launch CUDA kernel
	int n = pop.size();
	getWorthCudaKernel<<<1, n>>>(map);
	hipDeviceSynchronize(); //must remember this, or bad things will happen. baad things
	//cuda kernel launched and finished
	for (int i = 0; i < pop.size(); i++) {
		int temp;
		hipMemcpy(&temp, &map[i]->result, sizeof(int), hipMemcpyDeviceToHost);
		pop[i].worth = temp;
		for (auto &t : bitsets) freeBitsetArray((DeviceBitset**)t, pop[i].vertices.size());
	}
	for (auto &t : general) hipFree(t);
}