#include "hip/hip_runtime.h"
#include "../Headers/DeviceGraph.cuh"
#include "hip/hip_runtime.h"

__device__ int RyBKA(DeviceBitset *stack, int *map, int *rsstack, int N, const DeviceGraph *graph) {
	int stackIdx = 1, cmax = -1;
	while (stackIdx >= 0) {//while stack not empty
		stackIdx--; //stack pop
		if (stack[stackIdx].n == 0) {//if the P set is empty
			if (cmax < rsstack[stackIdx]) cmax = rsstack[stackIdx]; //check if found clique was greater than the previous one, if so, set
		}
		else {//if P is not empty
			int i = 0;//pick a vertex v
			while (!stack[stackIdx][i]) i++;//that exists in P
			for (int j = 0; j < N; j++) {//push (P \ v)
				if (stack[stackIdx][j] && j != i) stack[stackIdx + 1].set(i, 1);
				else stack[stackIdx + 1].set(i, 0);
			}
			rsstack[stackIdx + 1] = rsstack[stackIdx];
			stack[stackIdx + 1].n = stack[stackIdx].n - 1;
			stackIdx++;
			int m = 0;
			for (int j = 0; j < N; j++) {//for every other vertex
				if (stack[stackIdx][j]) {//that exists, do
					if (graph->isEdge(map[i], map[j])) {//check for edge by the way of map
						stack[stackIdx + 1].set(j, 1); //if connected, add to next iteration
						m++; //also, count
					}
					else stack[stackIdx + 1].set(j, 0); //if not, make sure it won't be there.
				}

			}
			stack[stackIdx + 1].n = m;
			rsstack[stackIdx + 1] = rsstack[stackIdx] + 1;
			stackIdx++;
		}
	}
	return cmax;
}

__device__ void getWorthDev(DeviceBKInput *in) {
	in->result = RyBKA(*in->set, in->map, in->rsstack, in->set[0]->n, in->g);
}
__host__ void indirectMalloc(void **ptr, int size) {
	void *p;
	hipMalloc(&p, size);
	hipMemcpy(ptr, &p, sizeof(void*), hipMemcpyHostToDevice);
}

__host__ DeviceGraph* loadGraphToDevice(const Graph *g) {
	DeviceGraph *res;
	hipMalloc((void**)&res, sizeof(DeviceGraph));
	indirectMalloc((void**)&res->vertices, g->vertexAmount*sizeof(DeviceGraphVertex));
	for (int i = 0; i < g->vertexAmount; i++) {
		DeviceGraphVertex v;
		v.neighbors = new int[g->vertices[i].neighbourhood.size()];
		int *neighbors;
		hipMalloc((void**)&neighbors, g->vertices[i].neighbourhood.size()*sizeof(int));
		for (unsigned int j = 0; j < g->vertices[i].neighbourhood.size(); j++) {
			v.neighbors[j] = g->vertices[i].neighbourhood[j];
		}
		v.degree = g->vertices[i].neighbourhood.size();
		hipMemcpy(neighbors, v.neighbors, g->vertices[i].neighbourhood.size()*sizeof(int), hipMemcpyHostToDevice);
		delete[] v.neighbors;
		v.neighbors = neighbors;
		hipMemcpy((&res->vertices)+i*sizeof(DeviceGraphVertex), &v, sizeof(DeviceGraphVertex), hipMemcpyHostToDevice);
	}
	hipMemcpy(&res->n, &g->vertexAmount, sizeof(int), hipMemcpyHostToDevice);
	return res;
}

__device__ bool DeviceBitset::operator[](int n) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = 0x1 << rem;
	return (contents[idx] & mask) ? true : false;
}

__device__ void DeviceBitset::set(int n, bool v) {
	int idx = n / 8;
	int rem = n % 8;
	char mask = (v ? 1 : 0) << rem;
	contents[idx] = contents[idx] | mask;
}

__host__ DeviceBitset* createDeviceBitset(int n) {
	DeviceBitset *res;
	hipMalloc((void**)&res, sizeof(DeviceBitset));
	int idx = n / 8;
	indirectMalloc((void**)&res->contents, sizeof(char)*idx);
	for (int i = 0; i < idx; i++) res->contents[i] = (signed char)0xFF;
	hipMemcpy(&res->n, &n, sizeof(int), hipMemcpyHostToDevice);
	return res;
}

__host__ DeviceBitset** createBitsetArray(int n) {
	DeviceBitset **res;
	hipMalloc((void**)&res, n*sizeof(DeviceBitset*));
	for (int i = 0; i < n; i++) {
		DeviceBitset *temp = createDeviceBitset(n);
		hipMemcpy((&res) + i*sizeof(DeviceBitset*), &temp, sizeof(DeviceBitset*), hipMemcpyHostToDevice);
	}
	return res;
}

__host__ void unloadDeviceGraph(DeviceGraph *g) {
	int n;
	hipMemcpy(&g->n, &n, sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++) {
		DeviceGraphVertex v;
		hipMemcpy(&g->vertices + i*sizeof(DeviceGraphVertex), &v, sizeof(DeviceGraphVertex), hipMemcpyDeviceToHost);
		hipFree(v.neighbors);
	}
	hipFree(g);
}
__host__ void freeDeviceBitset(DeviceBitset *b) {
	hipFree(b->contents);
	hipFree(b);
}

__host__ void freeBitsetArray(DeviceBitset **arr, int n) {
	for (int i = 0; i < n; i++) hipFree(arr+i*sizeof(DeviceBitset*));
	hipFree(arr);
}

__global__ void getWorthCudaKernel(DeviceBKInput **roadmap) {
	int myId = threadIdx.x;
	DeviceBKInput *myInput = roadmap[myId];
	getWorthDev(myInput);
}

__host__ void getWorthWithCuda(std::vector<Organism> &pop, DeviceGraph *g) {
	DeviceBKInput **map, **hostCopy = new DeviceBKInput*[pop.size()];
	std::vector<void*> general, bitsets, results;
	hipMalloc((void**)&map, sizeof(DeviceBKInput*)*pop.size());
	general.push_back(map);
	for (unsigned int i = 0; i < pop.size(); i++) {
		DeviceBKInput *in;
		hipMalloc((void**)&in, sizeof(DeviceBKInput));
		general.push_back(in);

		int *vertexMap;
		hipMalloc((void**)&vertexMap, pop[i].vertices.size()*sizeof(int));
		general.push_back(vertexMap);
		int *tempArray = new int[pop[i].vertices.size()], j=0;
		for (auto &t : pop[i].vertices) tempArray[j++] = t;
		hipMemcpy(vertexMap, tempArray, sizeof(int)*pop[i].vertices.size(), hipMemcpyHostToDevice);
		delete[] tempArray;
		hipMemcpy(&in->map, &vertexMap, sizeof(vertexMap), hipMemcpyHostToDevice);

		int *resstack;
		hipMalloc((void**)&resstack, pop[i].vertices.size()*sizeof(int));
		hipMemset(resstack, 0, pop[i].vertices.size()*sizeof(int));
		hipMemcpy(&in->rsstack, &resstack, sizeof(resstack), hipMemcpyHostToDevice);

		DeviceBitset** t = createBitsetArray(pop[i].vertices.size());
		hipMemcpy(&in->set, &t, sizeof(t), hipMemcpyHostToDevice);

		bitsets.push_back(t);
		results.push_back(&in->result);

		hipMemcpy(&in->g, &g, sizeof(g), hipMemcpyHostToDevice);
		hipMemcpy(&map[i], &in, sizeof(in), hipMemcpyHostToDevice);
	}
	//input array ready
	//time to launch CUDA kernel
	int n = pop.size();
	getWorthCudaKernel<<<1, n>>>(map);
	hipDeviceSynchronize(); //must remember this, or bad things will happen. baad things
	//cuda kernel launched and finished
	for (int i = 0; i < pop.size(); i++) {
		int temp;
		hipMemcpy(&temp, results[i], sizeof(int), hipMemcpyDeviceToHost);
		pop[i].worth = temp;
		for (auto &t : bitsets) freeBitsetArray((DeviceBitset**)t, pop[i].vertices.size());
	}
	for (auto &t : general) hipFree(t);
}